#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

#define SRAMSIZE 49152  // 49152, br = 384 or 192
#define NUMPAD 128
#define BR 32
#define BC 32

void input(char *input_filename);
void output(char *output_filename);
int ceil(int a, int b);

__global__ void dev_flash_attention(float* Q, float* K, float* V, float* O, float* l, float* m, int B, int N, int d, int bc, int br, int tc, int tr, float scaler);

float _max(float a, float b) { return a > b ? a : b; }
float _min(float a, float b) { return a < b ? a : b; }

double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

int B, N, d, embedding_dimension;
float *Q, *K, *V, *O, scaler;


int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }
    double start, end;
    start = getTimeStamp();

    input(argv[1]);



    // Initialize Q, K, V in HBM
    float *dev_Q, *dev_K, *dev_V;
    hipMalloc(&dev_Q, B * N * d * sizeof(float));
    hipMemcpy(dev_Q, Q, B * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&dev_K, B * N * d * sizeof(float));
    hipMemcpy(dev_K, K, B * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&dev_V, B * N * d * sizeof(float));
    hipMemcpy(dev_V, V, B * N * d * sizeof(float), hipMemcpyHostToDevice);

    // accumulation buffers (used for each batch)
    float *l = (float *)malloc(N * sizeof(float));  // Accumulates scaling factors
    float *m = (float *)malloc(N * sizeof(float));  // Tracks maximum values for numerical stability
    memset(l, 0x00, N * sizeof(float));
    for (int i = 0; i < N; i++) {
        m[i] = FLT_MIN;
    }
    // Step 2: Initialize O, l, m in HBM
    float *dev_O, *dev_l, *dev_m;
    hipMalloc(&dev_O, B * N * d * sizeof(float));
    hipMemcpy(dev_O, O, B * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&dev_l, B * N * sizeof(float));
    hipMemcpy(dev_l, l, B * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&dev_m, B * N * sizeof(float));
    hipMemcpy(dev_m, m, B * N * sizeof(float), hipMemcpyHostToDevice);

    // Step 1: Define block sizes for tiling
    // int br = SRAMSIZE / (4 * d), bc = d;  // br = 384 or 192, bc = 32 or 64, br * bc = M / 4 = 12288, ensure sufficient shared memory for 4 arrays, but no sufficient threads
    // max #threads = 1024, (1024 / 32, 32) = (32, 32), (1024 / 64, 64) = (16, 64)
    // int tr = ceil(N, br), tc = ceil(N, bc)
    // tr, tc: max_N = 32768, 32768 / 384 = 85.3 = 86, tc = 32768 / 32 = 1024
    int br = BR, bc = BC;
    int tr = N / br, tc = N / bc;  // tr = 4, 8, 16, 32, 64, 128, 256, 512, 1024 (32768 / 32), how many column for each set
    // dim3 threads(br, bc);          // (32, 32)
    dim3 threads(tc);
    dim3 blocks(B);
    // dim3 blocks(B, tr);   // (batch size, tr) -> only need tc times for each thread
    scaler = 1.0 / sqrt(d);

    // Calculate requested SRAM
    const int sram_size = (2 * bc * d * sizeof(float) + 1 * br * d * sizeof(float) + 1 * br * bc * sizeof(float));
    printf("requested shared memory: %d\n", sram_size);

    dev_flash_attention<<<blocks, threads, SRAMSIZE>>>(dev_Q, dev_K, dev_V, dev_O, dev_l, dev_m, B, N, d, bc, br, tc, tr, scaler);

    // for (int i = 0; i < B; i++) {
    //     flash_attention(
    //         Q + (i * N * d),
    //         K + (i * N * d),
    //         V + (i * N * d),
    //         O + (i * N * d)
    //     );
    // }

    end = getTimeStamp();
    // printf("(B, N, d): (%d, %d, %d)\n", B, N, embedding_dimension);
    printf("(B, N, d): (%d, %d, %d)\n", B, N, d);
    printf("Time: %.3f seconds\n", end - start);

    output(argv[2]);

    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    // fread(&embedding_dimension, sizeof(int), 1, file);
    // d = (embedding_dimension % NUMPAD) ? embedding_dimension : (embedding_dimension / NUMPAD + 1) * NUMPAD;
    fread(&d, sizeof(int), 1, file);

    Q = (float *)malloc(B * N * d * sizeof(float));
    K = (float *)malloc(B * N * d * sizeof(float));
    V = (float *)malloc(B * N * d * sizeof(float));
    O = (float *)malloc(B * N * d * sizeof(float));

    for (int i = 0; i < B; i++) {
        fread(Q + (i * N * d), sizeof(float), N * d, file);
        fread(K + (i * N * d), sizeof(float), N * d, file);
        fread(V + (i * N * d), sizeof(float), N * d, file);
    }
    memset(O, 0x00, B * N * d * sizeof(float));

    fclose(file);
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void dev_flash_attention(float* Q, float* K, float* V, float* O, float* l, float* m, int B, int N, int d, int bc, int br, int tc, int tr, float scaler) {
    if (blockIdx.x * N * d + threadIdx.x * bc >= N) {
        return;
    }

    int tc_start = threadIdx.x;
    // int bc_start = threadIdx.y;
    // int batch_start = blockIdx.x * blockDim.x;
    int batch_start = blockIdx.x;
    int kv_offset;
    int qo_offset;
    int lm_offset;

    // Step 3: Declare device variables for blocks of kj, vj, qi, and oi
    extern __shared__ float shared[];
    float *kj = shared;       // kj[d][bc]
    float* vj = kj + bc * d;  // vj[bc][d]
    float* qi = vj + bc * d;  // qi[br][d]
    float* s = qi + bc * d;   // s[br][bc]

    // Step 4: Declare device variables for blocks of li, mi, sij, pij, mij, lij, mi_new, li_new
    float *li = s + br * bc;  // li[br]
    float *mi = li + br;      // mi[br]
    // float sij;
    float pv;
    // float pij;
    float *mij = mi + br;   // mij[br]
    float *lij = mij + br;  // lij[br]
    float mi_new;
    float li_new;
    float sum;
    // for (int r = 0; r < br; ++r) {
    //     mi[r] = FLT_MIN;
    //     li[r] = FLT_MIN;
    // }

    // Outer loop: Iterate over blocks of K and V
    // for (int j = 0; j < tc; j++) {
        // Load block of K and V from HBM to SRAM
        // memcpy(kj, k + j * bc * d, bc * d * sizeof(float));
        // memcpy(vj, v + j * bc * d, bc * d * sizeof(float));
        kv_offset = (batch_start * N * d) + (tc_start * bc * d);

        for (int c = 0; c < bc; ++c) {
            for (int idx = 0; idx < d; ++idx) {
                kj[c * d + idx] = K[kv_offset + c * d + idx];
                vj[c * d + idx] = V[kv_offset + c * d + idx];
            }
        }
        __syncthreads();

        // Inner loop: Iterate over blocks of Q
        for (int i = 0; i < tr; i++) {
            // Load block of Q, O to SRAM and mi, li to registers
            // memcpy(qi, q + i * br * d, br * d * sizeof(float));
            // memcpy(oi, o + i * br * d, br * d * sizeof(float));
            // memcpy(li, l + i * br, br * sizeof(float));
            // memcpy(mi, m + i * br, br * sizeof(float));
            qo_offset = (batch_start * N * d) + (i * br * d);
            lm_offset = (batch_start * N) + (i * br);
            for (int r = 0; r < br; ++r) {
                for (int idx = 0; idx < d; ++idx) {
                    qi[r * d + idx] = Q[qo_offset + r * d + idx];
                    // oi[r * d + idx] = O[qo_offset + r * d + idx];
                }
                mi[r] = m[lm_offset + r];
                li[r] = l[lm_offset + r];
            }
            __syncthreads();
            // lm_offset = batch_start * N + i * br + bc_start;
            // mi = m[lm_offset];
            // li = l[lm_offset];

            // Compute scaled dot-product of Q and K blocks
            // QKDotAndScalar(sij, qi, kj, br, bc, 1.0 / sqrt(d));  // Kernel fusion
            // Softmax computation over the block
            // RowMax(mij, sij, br, bc);  // Find row-wise maximum for numerical stability
            for (int r = 0; r < br; ++r) {
                for (int c = 0; c < bc; ++c) {
                    // mij[r] = FLT_MIN;
                    sum = 0;
                    for (int idx = 0; idx < d; ++idx) {
                        sum += qi[r * d + idx] * kj[c * d + idx];
                    }
                    sum *= scaler;
                    s[(r * bc) + c] = sum;
                    // mij[r] = max(mij[r], sum);
                }
            }

            for (int r = 0; r < br; ++r) {
                mij[r] = s[r * bc];
                for (int c = 0; c < bc; ++c) {
                    mij[r] = max(mij[r], s[r * bc + c]);
                }
            }

            // MinusMaxAndExp(pij, sij, mij, br, bc);  // Subtract max and exponentiate
            // RowSum(lij, pij, br, bc);  // Compute row-wise sum for normalization
            for (int r = 0; r < br; ++r) {
                // lij[r] = 0;
                for (int c = 0; c < bc; ++c) {
                    s[(r * bc) + c] = expf(s[(r * bc) + c] - mij[r]);
                    // lij[r] += s[(r * bc) + c];
                }
            }
            for (int r = 0; r < br; ++r) {
                lij[r] = 0;
                for (int c = 0; c < bc; ++c) {
                    lij[r] += s[(r * bc) + c];
                }
            }

            // Update running max, sum, and output blocks
            // UpdateMiLiOi(mi, li, oi, mij, lij, pij, vj, br, bc);
            for (int r = 0; r < br; ++r) {
                mi_new = max(mi[r], mij[r]);
                li_new = (expf(mi[r] - mi_new) * li[r]) + (expf(mij[r] - mi_new) * lij[r]);
                for (int idx = 0; idx < d; ++idx) {
                    pv = 0.0F;
                    for (int c = 0; c < bc; ++c) {
                        pv += s[(r * bc) + c] * vj[c * d + idx];
                    }
                    // oi[i * d + j] = (li[i] * exp(mi[i] - mi_new[i]) * oi[i * d + j] + exp(mij[i] - mi_new[i]) * pv) / li_new[i];
                    O[qo_offset + r * d + idx] = ((li[r] * expf(mi[r] - mi_new) * O[qo_offset + r * d + idx]) + \
                                                  (expf(mij[r] - mi_new) * pv)) / li_new;
                }
                m[lm_offset + r] = mi_new;
                l[lm_offset + r] = li_new;
                // mi[r] = mi_new;
                // li[r] = li_new;
            }

            // Store updated output and normalization factors back to global memory
            // memcpy(o + i * br * d, oi, br * d * sizeof(float));
            // memcpy(l + i * br, li, br * sizeof(float));
            // memcpy(m + i * br, mi, br * sizeof(float));
        }
        __syncthreads();
    // }
}
